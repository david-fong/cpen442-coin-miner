#include "hip/hip_runtime.h"
/** Adapted from https://github.com/moffa13/SHA256CUDA */
#include <hip/hip_runtime.h>
#include <>

#include "sha256.cuh"

#include <iostream>
#include <iomanip>
#include <chrono>
#include <thread>
#include <string>
#include <cmath>
#include <cassert>
#include <cstring>


#define SHOW_INTERVAL_MS 10000
#define BLOCK_SIZE 256
#define SHA_PER_ITERATIONS 8'388'608
#define NUMBLOCKS (SHA_PER_ITERATIONS + BLOCK_SIZE - 1) / BLOCK_SIZE


static size_t difficulty = 1;

// Output string by the device read by host
uint8_t* g_nonce_out = nullptr;
uint8_t* g_hash_out = nullptr;
int* g_found = nullptr;

static uint64_t nonce = 0;
static uint64_t user_nonce = 0; // We don't need this because we have team_member_id
static uint64_t last_nonce_since_update = 0;

// Last timestamp we printed debug info
static std::chrono::high_resolution_clock::time_point t_last_updated;


void print_hex_bytes(std::ostream& os, const uint8_t* bytes, size_t bytes_size) {
	os << std::hex << std::setfill('0');
	for (uint8_t i = 0; i < bytes_size; ++i) {
		os << std::setw(2) << static_cast<int>(bytes[i]);
	}
	os << std::dec << std::endl;
}


__device__ uint8_t count_leading_zero_nibbles_(const uint8_t* const data, const uint8_t difficulty) {
	unsigned count = 0;
	unsigned i = 0;
	for (; i < difficulty/2; i++) {
		if (data[i]) [[likely]] {
			if (!(data[i] & 0xf0)) { count += 1; }
			return count;
		}
		else { count += 2; }
	}
	if (!(data[i] & 0xf0)) { count += 1; }
	return count;
}


__device__ uint8_t nonce_to_bytes(uint64_t nonce, uint8_t* out) {
	for (unsigned i = 0; i < 8; i++) {
		out[i] = static_cast<uint8_t>(nonce >> (8 * i));
	}
	return 8;
}


extern __shared__ uint8_t threads_buffer[];
__global__ void sha256_kernel(
	uint8_t* out_nonce, uint8_t* out_found_hash, int *out_found,
	const char* const prefix_str, const usize_t prefix_str_size,
	const uint8_t difficulty, const uint64_t nonce_seed,
	const char* miner_id_str // 32 bytes
) {
	const SHA256_CTX* const hasher_prefix_sha = &threads_buffer[0];
	uint8_t* const miner_id = &threads_buffer[sizeof(SHA256_CTX)];

	// If this is the first thread of the block, init the constants in shared memory
	if (threadIdx.x == 0) {
		sha256_init(&hasher_prefix);
		sha256_update(&hasher_prefix, prefix_str, prefix_str_size);
		memcpy(miner_id, miner_id_str, 32);
	}
	__syncthreads(); // Ensure the constants have been written to SMEM

	// Respects the memory padding of 8 bit (uint8_t).
	const size_t miner_threads_buffer = static_cast<size_t>(std::ceil((sizeof(SHA256_CTX) + 32 + 1) / 8.f) * 8);
	const uintptr_t md_addr = threadIdx.x * (64) + miner_threads_buffer;
	const uintptr_t nonce_addr = md_addr + 32;

	uint8_t* const md = &threads_buffer[md_addr];
	uint8_t* const nonce = &threads_buffer[nonce_addr];
	memset(nonce, 0, 32);
	nonce_to_bytes(nonce_seed + (blockIdx.x * blockDim.x + threadIdx.x), nonce);
	{
		SHA256_CTX hasher = *hasher_prefix;
		sha256_update(&hasher, nonce, 32);
		sha256_update(&hasher, miner_id, 32);
		sha256_final(&hasher, md);
	}
	if ((count_leading_zero_nibbles_(md, difficulty) >= difficulty) && (atomicExch(out_found, 1) == 0)) {
		memcpy(out_found_hash, md, 32);
		memcpy(out_nonce, nonce, 32);
	}
}


void print_state() {
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

	std::chrono::duration<double, std::milli> last_show_interval = t2 - t_last_updated;

	if (last_show_interval.count() > SHOW_INTERVAL_MS) {
		std::chrono::duration<double, std::milli> span = t2 - t_last_updated;
		float ratio = span.count() / 1000;
		std::clog << span.count() << " " << nonce - last_nonce_since_update << std::endl;
		std::clog << std::fixed << static_cast<uint64_t>((nonce - last_nonce_since_update) / ratio) << " hashes/s" << std::endl;
		std::clog << std::fixed << "nonce: " << nonce << std::endl;

		t_last_updated = std::chrono::high_resolution_clock::now();
		last_nonce_since_update = nonce;
	}
}


int main(const int argc, char const *const argv[]) {
	hipSetDevice(0);
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	t_last_updated = std::chrono::high_resolution_clock::now();

	const std::string id_of_miner(argv[1]);
	std::string team_member_id(argv[2]); team_member_id.resize(8, '\0');
	const std::string last_coin(argv[3]);
	difficulty = std::stoi(argv[4]);
	// num_threads (ignored)

	const std::string prefix_str = std::string("CPEN 442 Coin2021") + last_coin + team_member_id;

	// std::clog << "Nonce: ";
	// std::cin >> user_nonce;

	char* g_prefix_str = nullptr;
	hipMalloc(&g_prefix_str, prefix_str.size()+1);
	hipMemcpy(g_prefix_str, prefix_str.c_str(), prefix_str.size()+1, hipMemcpyHostToDevice);

	char* g_id_of_miner = nullptr;
	hipMalloc(&g_id_of_miner, id_of_miner.size()+1);
	hipMemcpy(g_id_of_miner, id_of_miner.c_str(), id_of_miner.size()+1, hipMemcpyHostToDevice);

	hipMallocManaged(&g_nonce_out, 32);
	hipMallocManaged(&g_hash_out, 32);
	hipMallocManaged(&g_found, sizeof(int));
	*g_found = 0;

	nonce += user_nonce;
	last_nonce_since_update += user_nonce;

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));

	const size_t dynamic_shared_size = (
		ceil((sizeof(SHA256_CTX)
		+ 32 // id_of_miner
		+ 1
	) / 8.f) * 8) + (64 * BLOCK_SIZE);
	std::clog << "Shared memory is " << dynamic_shared_size / 1024 << "KB" << std::endl;

	while (!*g_found) {
		sha256_kernel << < NUMBLOCKS, BLOCK_SIZE, dynamic_shared_size >> > (
			g_nonce_out, g_hash_out, g_found,
			g_prefix_str, prefix_str.size(),
			difficulty, nonce,
			g_id_of_miner, id_of_miner.size()
		);
		hipError_t err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			throw std::runtime_error("Device error");
		}
		nonce += NUMBLOCKS * BLOCK_SIZE;
		print_state();
	}
	print_hex_bytes(std::clog, g_hash_out, 32);

	// coin_blob:
	print_hex_bytes(std::cout, (const uint8_t*)team_member_id.data(), team_member_id.size());
	print_hex_bytes(std::cout, g_nonce_out, 32);

	hipFree(g_nonce_out);
	hipFree(g_hash_out);
	hipFree(g_found);
	hipFree(g_id_of_miner);
	hipDeviceReset();
	return 0;
}